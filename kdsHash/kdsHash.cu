#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <stdint.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const uint crc32df[256] = {
    0x00000000, 0x77073096, 0xEE0E612C, 0x990951BA,
    0x076DC419, 0x706AF48F, 0xE963A535, 0x9E6495A3,
    0x0EDB8832, 0x79DCB8A4, 0xE0D5E91E, 0x97D2D988,
    0x09B64C2B, 0x7EB17CBD, 0xE7B82D07, 0x90BF1D91,
    0x1DB71064, 0x6AB020F2, 0xF3B97148, 0x84BE41DE,
    0x1ADAD47D, 0x6DDDE4EB, 0xF4D4B551, 0x83D385C7,
    0x136C9856, 0x646BA8C0, 0xFD62F97A, 0x8A65C9EC,
    0x14015C4F, 0x63066CD9, 0xFA0F3D63, 0x8D080DF5,
    0x3B6E20C8, 0x4C69105E, 0xD56041E4, 0xA2677172,
    0x3C03E4D1, 0x4B04D447, 0xD20D85FD, 0xA50AB56B,
    0x35B5A8FA, 0x42B2986C, 0xDBBBC9D6, 0xACBCF940,
    0x32D86CE3, 0x45DF5C75, 0xDCD60DCF, 0xABD13D59,
    0x26D930AC, 0x51DE003A, 0xC8D75180, 0xBFD06116,
    0x21B4F4B5, 0x56B3C423, 0xCFBA9599, 0xB8BDA50F,
    0x2802B89E, 0x5F058808, 0xC60CD9B2, 0xB10BE924,
    0x2F6F7C87, 0x58684C11, 0xC1611DAB, 0xB6662D3D,
    0x76DC4190, 0x01DB7106, 0x98D220BC, 0xEFD5102A,
    0x71B18589, 0x06B6B51F, 0x9FBFE4A5, 0xE8B8D433,
    0x7807C9A2, 0x0F00F934, 0x9609A88E, 0xE10E9818,
    0x7F6A0DBB, 0x086D3D2D, 0x91646C97, 0xE6635C01,
    0x6B6B51F4, 0x1C6C6162, 0x856530D8, 0xF262004E,
    0x6C0695ED, 0x1B01A57B, 0x8208F4C1, 0xF50FC457,
    0x65B0D9C6, 0x12B7E950, 0x8BBEB8EA, 0xFCB9887C,
    0x62DD1DDF, 0x15DA2D49, 0x8CD37CF3, 0xFBD44C65,
    0x4DB26158, 0x3AB551CE, 0xA3BC0074, 0xD4BB30E2,
    0x4ADFA541, 0x3DD895D7, 0xA4D1C46D, 0xD3D6F4FB,
    0x4369E96A, 0x346ED9FC, 0xAD678846, 0xDA60B8D0,
    0x44042D73, 0x33031DE5, 0xAA0A4C5F, 0xDD0D7CC9,
    0x5005713C, 0x270241AA, 0xBE0B1010, 0xC90C2086,
    0x5768B525, 0x206F85B3, 0xB966D409, 0xCE61E49F,
    0x5EDEF90E, 0x29D9C998, 0xB0D09822, 0xC7D7A8B4,
    0x59B33D17, 0x2EB40D81, 0xB7BD5C3B, 0xC0BA6CAD,
    0xEDB88320, 0x9ABFB3B6, 0x03B6E20C, 0x74B1D29A,
    0xEAD54739, 0x9DD277AF, 0x04DB2615, 0x73DC1683,
    0xE3630B12, 0x94643B84, 0x0D6D6A3E, 0x7A6A5AA8,
    0xE40ECF0B, 0x9309FF9D, 0x0A00AE27, 0x7D079EB1,
    0xF00F9344, 0x8708A3D2, 0x1E01F268, 0x6906C2FE,
    0xF762575D, 0x806567CB, 0x196C3671, 0x6E6B06E7,
    0xFED41B76, 0x89D32BE0, 0x10DA7A5A, 0x67DD4ACC,
    0xF9B9DF6F, 0x8EBEEFF9, 0x17B7BE43, 0x60B08ED5,
    0xD6D6A3E8, 0xA1D1937E, 0x38D8C2C4, 0x4FDFF252,
    0xD1BB67F1, 0xA6BC5767, 0x3FB506DD, 0x48B2364B,
    0xD80D2BDA, 0xAF0A1B4C, 0x36034AF6, 0x41047A60,
    0xDF60EFC3, 0xA867DF55, 0x316E8EEF, 0x4669BE79,
    0xCB61B38C, 0xBC66831A, 0x256FD2A0, 0x5268E236,
    0xCC0C7795, 0xBB0B4703, 0x220216B9, 0x5505262F,
    0xC5BA3BBE, 0xB2BD0B28, 0x2BB45A92, 0x5CB36A04,
    0xC2D7FFA7, 0xB5D0CF31, 0x2CD99E8B, 0x5BDEAE1D,
    0x9B64C2B0, 0xEC63F226, 0x756AA39C, 0x026D930A,
    0x9C0906A9, 0xEB0E363F, 0x72076785, 0x05005713,
    0x95BF4A82, 0xE2B87A14, 0x7BB12BAE, 0x0CB61B38,
    0x92D28E9B, 0xE5D5BE0D, 0x7CDCEFB7, 0x0BDBDF21,
    0x86D3D2D4, 0xF1D4E242, 0x68DDB3F8, 0x1FDA836E,
    0x81BE16CD, 0xF6B9265B, 0x6FB077E1, 0x18B74777,
    0x88085AE6, 0xFF0F6A70, 0x66063BCA, 0x11010B5C,
    0x8F659EFF, 0xF862AE69, 0x616BFFD3, 0x166CCF45,
    0xA00AE278, 0xD70DD2EE, 0x4E048354, 0x3903B3C2,
    0xA7672661, 0xD06016F7, 0x4969474D, 0x3E6E77DB,
    0xAED16A4A, 0xD9D65ADC, 0x40DF0B66, 0x37D83BF0,
    0xA9BCAE53, 0xDEBB9EC5, 0x47B2CF7F, 0x30B5FFE9,
    0xBDBDF21C, 0xCABAC28A, 0x53B39330, 0x24B4A3A6,
    0xBAD03605, 0xCDD70693, 0x54DE5729, 0x23D967BF,
    0xB3667A2E, 0xC4614AB8, 0x5D681B02, 0x2A6F2B94,
    0xB40BBE37, 0xC30C8EA1, 0x5A05DF1B, 0x2D02EF8D
};

typedef struct
{
    uint    cid;
    uint    mat;
} dti;

typedef struct
{
    uint    nid;
    float   xid;
    float   yid;
    float   zid;
} kti;

typedef struct
{
    uint    hash;
    uint    next;
    uint    rowid;
    uint    __padding__;
    kti     t;
} khi;

typedef struct {
    uint    crc32t[256];
    uint    length;
    uint    ncols;
    uint    nitems;
    uint    nslots;
    uint    hash_min;
    uint    hash_max;
} kds;

#define get_local_id()      (threadIdx.x)
#define get_local_size()    (blockDim.x)
#define get_global_id()     (threadIdx.x + blockIdx.x * blockDim.x)
#define get_global_size()   (blockDim.x * gridDim.x)
#define get_global_base()   (blockIdx.x * blockDim.x)
#define get_global_index()  (blockIdx.x)

#define TYPEALIGN(ALIGNVAL,LEN) (((uint) (LEN) + ((ALIGNVAL) - 1)) & ~((uint) ((ALIGNVAL) - 1)))

#define INIT_LEGACY_CRC32(crc) ((crc)  = 0xFFFFFFFF)
#define FIN_LEGACY_CRC32(crc)  ((crc) ^= 0xFFFFFFFF)
#define COMP_LEGACY_CRC32(crc, data, len) COMP_CRC32_REFLECTED_TABLE(crc, data, len, crc32df)
#define EQ_LEGACY_CRC32(c1, c2) ((c1) == (c2))

#define COMP_CRC32_REFLECTED_TABLE(crc, data, len, table) \
do { \
    const unsigned char *__data = (const unsigned char *) (data); \
    uint __len = (len); \
    while (__len-- > 0) \
    { \
        int __tab_index = ((int) ((crc) >> 24) ^ *__data++) & 0xFF; \
        (crc) = table[__tab_index] ^ ((crc) << 8); \
    } \
} while (0)

#define KERN_DATA_STORE_HEAD_LENGTH(kds) TYPEALIGN(16,sizeof(kds))
#define KERN_DATA_STORE_BODY(kds) ((char *)(kds) + KERN_DATA_STORE_HEAD_LENGTH(kds))
#define KERN_DATA_STORE_ROWINDEX(kds) ((uint *)(KERN_DATA_STORE_BODY(kds)))
#define KERN_DATA_STORE_HASHSLOT(kds) ((uint *)(KERN_DATA_STORE_BODY(kds) + TYPEALIGN(16,100000 * sizeof(uint))))

static inline uint HOST_HASH_VALUE(int key)
{
    uint hash;
    INIT_LEGACY_CRC32(hash);
    COMP_LEGACY_CRC32(hash, &key, sizeof(int));
    FIN_LEGACY_CRC32(hash);
    return hash;
}

__device__ __forceinline__ static khi* KERN_HASH_FIRST_ITEM(kds *k, uint h)
{
    uint *slot = (uint *)(((char *)(k) + KERN_DATA_STORE_HEAD_LENGTH(kds)) + TYPEALIGN(16,100000 * sizeof(uint)));
    uint i = h % 4096;

    if (slot[i] == 0)
        return NULL;
    return (khi *)((char *)k + slot[i]);
}

__device__ __forceinline__ static khi* KERN_HASH_NEXT_ITEM(kds *k, khi *i)
{
    if (!i || i->next == 0)
        return NULL;
    return (khi *)((char *)k + i->next);
}

__global__ void gpu_hash_join(dti *dlhs, kds *drhs)
{
    uint i = 0,hash = 0,cid = 0;
    __shared__ uint crc32t[256];
    for (i = get_local_id(); i < 256; i += get_local_size())
    {
        crc32t[i] = drhs->crc32t[i];
    }
    __syncthreads();

    cid = dlhs[get_global_id()].cid;
    INIT_LEGACY_CRC32(hash);
    COMP_CRC32_REFLECTED_TABLE(hash, &cid, sizeof(int), crc32t);
    FIN_LEGACY_CRC32(hash);

    khi *item = KERN_HASH_FIRST_ITEM(drhs, hash);
    do
    {
        if (item && (item->hash == hash) && (cid == item->t.nid))
        {
            dlhs[get_global_id()].mat = 1;
        }
        item = KERN_HASH_NEXT_ITEM(drhs, item);
    } while (item != NULL);
}

int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);
    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);
    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    kds *rhs = NULL;
    uint rsize = TYPEALIGN(16,sizeof(kds)) + (100000 + 4096) * sizeof(uint) + 100000 * sizeof(khi);
    checkCudaErrors(hipHostMalloc((void **)&rhs, rsize));
    memset(rhs, 0, sizeof(kds));
    for (uint i = 0;i < 256;i++)
    {
        rhs->crc32t[i] = crc32df[i];
    }

    rhs->length = rsize;
    rhs->ncols = 4;
    rhs->nitems = 100000;
    rhs->nslots = 4096;
    rhs->hash_min = 0;
    rhs->hash_max = UINT_MAX;

    uint *row_index = (uint *)(((char *)(rhs) + KERN_DATA_STORE_HEAD_LENGTH(kds)));
    uint *hash_slot = (uint *)(((char *)(rhs) + KERN_DATA_STORE_HEAD_LENGTH(kds)) + TYPEALIGN(16,100000 * sizeof(uint)));
    memset(row_index, 0, (100000 + 4096) * sizeof(uint));

    khi *row;
    uint i,j,k;
    for (i = 0;i < 100000;i++)
    {
        row = (khi *)((char *)rhs + TYPEALIGN(16,sizeof(kds)) + (100000 + 4096) * sizeof(uint) + i * sizeof(khi));
        row->hash = HOST_HASH_VALUE(i);
        row->rowid = i;
        row->t.nid = i;
        row->t.xid = (float)(i+1);
        row->t.yid = (float)(i+2);
        row->t.zid = (float)(i+3);
        row_index[row->rowid] = (uint)((uintptr_t)&row->t - (uintptr_t)rhs);
        j = row->hash % 4096;
        row->next = hash_slot[j];
        hash_slot[j] = (uintptr_t)row - (uintptr_t)rhs;
    }

    int n = 128 * 1024 * 1024;
    int nbytes = n * sizeof(dti);
    // allocate host memory
    dti *lhs = NULL;
    checkCudaErrors(hipHostMalloc((void **)&lhs, nbytes));
    for (k = 0;k < n;k++)
    {
        lhs[k].cid = k;
        lhs[k].mat = 0;
    }

    // allocate device memory
    dti *dlhs = NULL;
    checkCudaErrors(hipMalloc((void **)&dlhs, nbytes));
    checkCudaErrors(hipMemset(dlhs, 0, nbytes));
    kds *drhs = NULL;
    checkCudaErrors(hipMalloc((void **)&drhs, rsize));
    checkCudaErrors(hipMemset(drhs, 0, rsize));

    // set kernel launch configuration
    dim3 threads = dim3(1024, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;
    // asynchronously issue work to the GPU (all to stream 0)
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(drhs, rhs, rsize, hipMemcpyHostToDevice, 0);
    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpyAsync(dlhs, lhs, nbytes, hipMemcpyHostToDevice, 0);
    gpu_hash_join<<<blocks, threads, 0, 0>>>(dlhs, drhs);
    hipMemcpyAsync(lhs, dlhs, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    int cnt = 0;
    for (uint i = 0;i < n;i++)
    {
        if (lhs[i].mat == 1)
            cnt += 1;
    }
    printf("all match item num: %d\n", cnt);

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(lhs));
    checkCudaErrors(hipHostFree(rhs));
    checkCudaErrors(hipFree(dlhs));
    checkCudaErrors(hipFree(drhs));

    return EXIT_SUCCESS;
}
